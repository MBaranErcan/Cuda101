#include "hip/hip_runtime.h"
﻿/*
These example works here are intended to implement the book "CUDA by Example" by Jason Sanders and Edward Kandrot.
*/

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <>

#include <Windows.h>

using namespace std;

#define N 10

// Function prototypes
bool isKeyPressed(unsigned timeout_ms);
void printCudaDeviceProperties();

__global__ void add(int* a, int* b, int* c);

int main(void) {
	
	// Print the Cuda device properties
	printCudaDeviceProperties();

	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	// Allocate memory on the device.
	hipMalloc((void**) &dev_a, N * sizeof(int));
	hipMalloc((void**) &dev_b, N * sizeof(int));
	hipMalloc((void**) &dev_c, N * sizeof(int));


	// Fİll the arrays a and b
	for (int i = 0; i < N; i++) {
		a[i] = 2-i;
		b[i] = i * i * i;
	}

	// Copy the arrays to device
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);


	add<<<N, 1>>>(dev_a, dev_b, dev_c);


	// Copy the result back to host
	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);


	// Print the result
	for (int i = 0; i < N; i++) {
		printf("a[%d]:%d + b[%d]:%d = c[%d]:%d\n", i, a[i], i, b[i], i, c[i]);
	}

	// Free the memory
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);


	// Program end
	printf("Press any key to escape...");
	while (!isKeyPressed(0)); // Wait for 10 seconds.

	return 0;
}


// Kernel function add two arrays
__global__ void add(int* a, int* b, int* c) {
	int tid = blockIdx.x;
	if (tid < N) c[tid] = a[tid] + b[tid];

}

// Check if a key is pressed, with a default timeout of 0 ms. 
// If key is pressed return true, if not wait auntil timeout and send false. 
// This is windows specific.
bool isKeyPressed(unsigned timeout_ms = 0)
{
	return WaitForSingleObject(
		GetStdHandle(STD_INPUT_HANDLE),
		timeout_ms
	) == WAIT_OBJECT_0;
}


// Print the device properties
void printCudaDeviceProperties() {
	hipDeviceProp_t prop;

	int count, i;
	hipGetDeviceCount(&count);

	for (i = 0; i < count; i++) {
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("Device name: %s\n", prop.name);
		printf("Compute capability: %d.%d\n", prop.major, prop.minor);
		printf("Clock rate: %d\n", prop.clockRate);
		printf("Device copy overlap: ");
		if (prop.deviceOverlap)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("Kernel execution timeout: ");
		if (prop.kernelExecTimeoutEnabled)
			printf("Enabled\n");
		else
			printf("Disabled\n");

		printf("   --- Memory Information for device %d ---\n", i);
		printf("Total global mem: %Iu\n", prop.totalGlobalMem); // Hint: to print size_t correctly, use %Iu, in windows.
		printf("Total constant mem: %Iu\n", prop.totalConstMem);
		printf("Max mem pitch: %Iu\n", prop.memPitch);
		printf("Texture Alignment: %Iu\n", prop.textureAlignment);

		printf("   --- MP Information for device: %d ---\n", i);
		printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
		printf("Shared mem per mp: %Iu\n", prop.sharedMemPerBlock);
		printf("Registers per mp: %d\n", prop.regsPerBlock);
		printf("Threads in warp: %d\n", prop.warpSize);
		printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
		printf("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("\n");
	}
}